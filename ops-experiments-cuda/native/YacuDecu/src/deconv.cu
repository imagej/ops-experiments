#include "hip/hip_runtime.h"
/*
    deconv.cu

    Author: Bob Pepin - (originally obtained from https://github.com/bobpepin/YacuDecu)
    Author: Brian Northan 
		- changes to dimension order of FFT plan in deconv_device function in order for this function to work on arrays from imglib2.
		- changed multiplication if Richardson Lucy loop to
		- Add convolution function
		- Add optional non-circulant normalization factor for edge handling  
			see (http://bigwww.epfl.ch/deconvolution/challenge/index.html?p=documentation/theory/richardsonlucy)

    License: LGPL

*/

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "deconv.h"

__global__ void ComplexMul(hipComplex *A, hipComplex *B, hipComplex *C)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
    C[i] = hipCmulf(A[i], B[i]);
}

// BN 2018 add complex conjugate multiply
__host__ __device__ static __inline__ hipFloatComplex cuCconjmulf(hipFloatComplex x,
	hipFloatComplex y)
{
	hipFloatComplex prod;
	prod = make_hipFloatComplex((hipCrealf(x) * hipCrealf(y)) +
		(hipCimagf(x) * hipCimagf(y)),
		-(hipCrealf(x) * hipCimagf(y)) +
		(hipCimagf(x) * hipCrealf(y)));
	return prod;
}

// BN 2018 add complex conjugate multiply kernel
__global__ void ComplexConjugateMul(hipComplex *A, hipComplex *B, hipComplex *C)
{
	unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
	C[i] = cuCconjmulf(A[i], B[i]);
}

__global__ void FloatDiv(float *A, float *B, float *C)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
    
	if (B[i] != 0) {
		C[i] = A[i] / B[i];
	}
	else {
		C[i] = 0;
	}

}

__global__ void FloatMul(float *A, float *B, float *C)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
	
    C[i] = A[i] * B[i];
}

__global__ void FloatDivByConstant(float *A, float constant)
{
    unsigned int i = blockIdx.x * gridDim.y * gridDim.z * blockDim.x + blockIdx.y * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
    A[i]=A[i]/constant;
}

static hipfftResult createPlans(size_t, size_t, size_t, hipfftHandle *planR2C, hipfftHandle *planC2R, void **workArea, size_t *workSize);
static hipError_t numBlocksThreads(unsigned int N, dim3 *numBlocks, dim3 *threadsPerBlock);

static float floatMean(float *a, int N) {
    float m = 0;
    for(float *p = a; p < a+N; p++) {
        m += *p;
    }
    return m / (float)N;
}

static float devFloatMean(float *a_dev, int N) {
    float *a = (float*)malloc(N*sizeof(float));
    hipMemcpy(a, a_dev, N*sizeof(float), hipMemcpyDeviceToHost);
    float m = floatMean(a, N);
    free(a);
    return m;
}

/* h_normal is the non-circulant normalization factor described here
	http://bigwww.epfl.ch/deconvolution/challenge/index.html?p=documentation/theory/richardsonlucyi
*/
int deconv_device(unsigned int iter, size_t N1, size_t N2, size_t N3, 
                  float *h_image, float *h_psf, float *h_object, float *h_normal) {
    int retval = 0;
    hipfftResult r;
    hipError_t err;
    hipfftHandle planR2C, planC2R;

	std::cout<<"Starting Cuda deconvolution\n";

    float *image = 0; // convolved image (constant)
    float *object = 0; // estimated object
	float *psf=0;
	float*temp=0;
	float*normal = 0;

    hipComplex *otf = 0; // Fourier transform of PSF (constant)
    void *buf = 0; // intermediate results
    void *workArea = 0; // cuFFT work area

    size_t nSpatial = N1*N2*N3; // number of values in spatial domain
    size_t nFreq = N1*N2*(N3/2+1); // number of values in frequency domain
    //size_t nFreq = N1*(N2/2+1); // number of values in frequency domain
    size_t mSpatial, mFreq;

    dim3 freqThreadsPerBlock, spatialThreadsPerBlock, freqBlocks, spatialBlocks;
    size_t workSize; // size of cuFFT work area in bytes

    err = numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock);
    if(err) goto cudaErr;
    err = numBlocksThreads(nFreq, &freqBlocks, &freqThreadsPerBlock);
    if(err) goto cudaErr;

    mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
    mFreq = freqBlocks.x * freqBlocks.y * freqBlocks.z * freqThreadsPerBlock.x * sizeof(hipComplex);

    printf("N: %ld, M: %ld\n", nSpatial, mSpatial);
    printf("Blocks: %d x %d x %d, Threads: %d x %d x %d\n", spatialBlocks.x, spatialBlocks.y, spatialBlocks.z, spatialThreadsPerBlock.x, spatialThreadsPerBlock.y, spatialThreadsPerBlock.z);
	fflush(stdin);

	std::cout<<"N: "<<nSpatial<<" M: "<<mSpatial<<"\n"<<std::flush;
	std::cout<<"Blocks: "<<spatialBlocks.x<<" x "<<spatialBlocks.y<<" x "<<spatialBlocks.z<<", Threads: "<<spatialThreadsPerBlock.x<<" x "<<spatialThreadsPerBlock.y<<" x "<<spatialThreadsPerBlock.z<<"\n";
    
	hipDeviceReset();
    hipProfilerStart();

    err = hipMalloc(&image, mSpatial);
    if(err) goto cudaErr;
    err = hipMalloc(&object, mSpatial);
    if(err) goto cudaErr;
	err = hipMalloc(&psf, mSpatial);
    if(err) goto cudaErr;
	//err = hipMalloc(&temp, mSpatial);
    //if(err) goto cudaErr;

	if (h_normal!=NULL) {
		err = hipMalloc(&normal, mSpatial);
		if (err) goto cudaErr;
	}
	else {
		normal = NULL;
	}

    err = hipMalloc(&otf, mFreq);
    if(err) goto cudaErr;
    err = hipMalloc(&buf, mFreq); // mFreq > mSpatial
    if(err) goto cudaErr;
	

    err = hipMemset(image, 0, mSpatial);
    if(err) goto cudaErr;
    err = hipMemset(object, 0, mSpatial);
    if(err) goto cudaErr;

	if (h_normal != NULL) {
		err = hipMemset(normal, 0, mSpatial);
		if (err) goto cudaErr;
	}

    err = hipMemcpy(image, h_image, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;

    err = hipMemcpy(object, h_object, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;

    err = hipMemcpy(psf, h_psf, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;

	if (h_normal != NULL) {
		err = hipMemcpy(normal, h_normal, nSpatial * sizeof(float), hipMemcpyHostToDevice);
		if (err) goto cudaErr;
	}


    // BN it looks like this function was originall written for the array organization used in matlab.  I Changed the order of the dimensions
    // to be compatible with imglib2 (java). TODO - add param for array organization 
    r = createPlans(N1, N2, N3, &planR2C, &planC2R, &workArea, &workSize);
    if(r) goto cufftError;

    printf("Plans created.\n");

    r = hipfftExecR2C(planR2C, psf, otf);
    if(r) goto cufftError;

	// since we don't the psf anymore (we just used it to get the OTF) use the psf buffer
	// as the temp buffer
	temp = psf;

    for(unsigned int i=0; i < iter; i++) {
        // BN flush the buffer for debugging in Java.
        fflush(stdout);
        
		std::cout<<"Iteration "<<i<<"\n"<<std::flush;

		r = hipfftExecR2C(planR2C, object, (hipfftComplex*)buf);
        if(r) goto cufftError;
        
		ComplexMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
        r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)temp);
        if(r) goto cufftError;
		FloatDivByConstant<<<spatialBlocks, spatialThreadsPerBlock>>>((float*)temp,(float)nSpatial);
		
        FloatDiv<<<spatialBlocks, spatialThreadsPerBlock>>>(image, (float*)temp, (float*)temp);
        
		r = hipfftExecR2C(planR2C, (float*)temp, (hipfftComplex*)buf);
        if(r) goto cufftError;

		// BN 2018 Changed to complex conjugate multiply
        ComplexConjugateMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
		r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)temp);
		if(r) goto cufftError;

		FloatDivByConstant<<<spatialBlocks, spatialThreadsPerBlock>>>((float*)temp,(float)nSpatial);
		
        FloatMul<<<spatialBlocks, spatialThreadsPerBlock>>>((float*)temp, object, object);
		
		if (normal != NULL) {
			FloatDiv<<<spatialBlocks, spatialThreadsPerBlock >>>((float*)object, normal, object);
		}
		
    }

	err = hipMemcpy(h_object, object, nSpatial*sizeof(float), hipMemcpyDeviceToHost);
    if(err) goto cudaErr;

    retval = 0;
    goto cleanup;

cudaErr:
    fprintf(stderr, "CUDA error: %d\n", err);
	std::cout << "CUDA error: " << err << std::endl;
	
    retval = err;
    goto cleanup;

cufftError:
    fprintf(stderr, "CuFFT error IS: %d\n", r);
	std::cout << "CuFFT error is: " << r << std::endl;

    retval = r;
    goto cleanup;

cleanup:
    if(image) hipFree(image);
    if(object) hipFree(object);
    if(otf) hipFree(otf);
    if(buf) hipFree(buf);
    if(workArea) hipFree(workArea);
    hipProfilerStop();
    hipDeviceReset();
    return retval;
}

extern "C" int deconv_host(unsigned int iter, size_t N1, size_t N2, size_t N3, 
                float *h_image, float *h_psf, float *h_object, float *h_normal) {
    int retval = 0;
    hipfftResult r;
    hipError_t err;
    hipfftHandle planR2C, planC2R;

    float *image = 0; // convolved image (constant)
    float *object = 0; // estimated object
    hipComplex *otf = 0; // Fourier transform of PSF (constant)
    void *buf = 0; // intermediate results
    void *workArea = 0; // cuFFT work area
    hipComplex *h_otf = 0;
    void *h_buf = 0;

    float *h_image_pad = 0;
    float *h_object_pad = 0;

    size_t nSpatial = N1*N2*N3; // number of values in spatial domain
    size_t nFreq = N1*N2*(N3/2+1); // number of values in frequency domain
    //size_t nFreq = N1*(N2/2+1); // number of values in frequency domain
    size_t mSpatial, mFreq;

    dim3 freqThreadsPerBlock, spatialThreadsPerBlock, freqBlocks, spatialBlocks;
    size_t workSize; // size of cuFFT work area in bytes

    err = numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock);
    if(err) goto cudaErr;
    err = numBlocksThreads(nFreq, &freqBlocks, &freqThreadsPerBlock);
    if(err) goto cudaErr;

    mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
    mFreq = freqBlocks.x * freqBlocks.y * freqBlocks.z * freqThreadsPerBlock.x * sizeof(hipComplex);

    printf("N: %ld, M: %ld\n", nSpatial, mSpatial);
    printf("Blocks: %d x %d x %d, Threads: %d x %d x %d\n", spatialBlocks.x, spatialBlocks.y, spatialBlocks.z, spatialThreadsPerBlock.x, spatialThreadsPerBlock.y, spatialThreadsPerBlock.z);

    hipDeviceReset();
    err = hipSetDeviceFlags(hipDeviceMapHost);
    printf("Set Device Flags: %d\n", err);

    hipProfilerStart();

    err = hipHostAlloc(&h_otf, mFreq, hipHostMallocMapped | hipHostMallocWriteCombined);
    if(err) goto cudaErr;
    err = hipHostAlloc(&h_buf, mFreq, hipHostMallocMapped | hipHostMallocWriteCombined);
    if(err) goto cudaErr;

    printf("Host memory allocated.\n");

    if(mSpatial > nSpatial*sizeof(float)) {
        err = hipHostAlloc(&h_image_pad, mSpatial, hipHostMallocMapped | hipHostMallocWriteCombined);
        if(err) goto cudaErr;
        err = hipHostAlloc(&h_object_pad, mSpatial, hipHostMallocMapped | hipHostMallocWriteCombined);
        if(err) goto cudaErr;
        err = hipHostGetDevicePointer(&image, h_image_pad, 0);
        if(err) goto cudaErr;
        err = hipHostGetDevicePointer(&object, h_object_pad, 0);
        if(err) goto cudaErr;
        err = hipMemcpy(image, h_image, nSpatial*sizeof(float), hipMemcpyHostToDevice);
        if(err) goto cudaErr;
        err = hipMemcpy(object, h_object, nSpatial*sizeof(float), hipMemcpyHostToDevice);
        if(err) goto cudaErr;
    } else {
        err = hipHostRegister(h_image, mSpatial, hipHostRegisterMapped);
        if(err) goto cudaErr;
        err = hipHostRegister(h_object, mSpatial, hipHostRegisterMapped);
        if(err) goto cudaErr;
        err = hipHostGetDevicePointer(&image, h_image, 0);
        if(err) goto cudaErr;
        err = hipHostGetDevicePointer(&object, h_object, 0);
        if(err) goto cudaErr;
    }
    err = hipHostGetDevicePointer(&otf, h_otf, 0);
    if(err) goto cudaErr;
    err = hipHostGetDevicePointer(&buf, h_buf, 0);
    if(err) goto cudaErr;

    printf("Host pointers registered.\n");

    err = hipMemcpy(otf, h_psf, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;
    printf("PSF transferred.\n");

    r = createPlans(N1, N2, N3, &planR2C, &planC2R, &workArea, &workSize);
    if(r) goto cufftError;

    printf("Plans created.\n");

    r = hipfftExecR2C(planR2C, (float*)otf, otf);
    if(r) goto cufftError;

    for(unsigned int i=0; i < iter; i++) {
        printf("Iteration %d\n", i);
        r = hipfftExecR2C(planR2C, object, (hipfftComplex*)buf);
        if(r) goto cufftError;
        ComplexMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
        r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)buf);
        if(r) goto cufftError;

        printf("a: m = %f\n", devFloatMean((float*)buf, nSpatial));
        FloatDiv<<<spatialBlocks, spatialThreadsPerBlock>>>(image, (float*)buf, (float*)buf);
        r = hipfftExecR2C(planR2C, (float*)buf, (hipfftComplex*)buf);
        if(r) goto cufftError;
        ComplexMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
        r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)buf);
        if(r) goto cufftError;
        FloatMul<<<spatialBlocks, spatialThreadsPerBlock>>>((float*)buf, object, object);
    }

    printf("object: m = %f\n", devFloatMean((float*)object, nSpatial));

    err = hipMemcpy(h_object, object, nSpatial*sizeof(float), hipMemcpyDeviceToHost);
    if(err) goto cudaErr;

    retval = 0;
    goto cleanup;

cudaErr:
    fprintf(stderr, "CUDA error: %d\n", err);
    retval = err;
    goto cleanup;

cufftError:
    fprintf(stderr, "CuFFT error: %d\n", r);
    retval = r;
    goto cleanup;

cleanup:
    printf("h_image: %p, h_object: %p, h_psf: %p, h_buf: %p, h_otf: %p\n", h_image, h_object, h_psf, h_buf, h_otf);
    if(image) {
        if(h_image_pad) {
            hipHostUnregister(h_image_pad);
            hipHostFree(h_image_pad);
        } else {
            hipHostUnregister(h_image);
        }
    }
    if(object) {
        if(h_object_pad) {
            hipHostUnregister(h_object_pad);
            hipHostFree(h_object_pad);
        } else {
            hipHostUnregister(h_object);
        }
    }
    if(otf) {
        hipHostUnregister(h_otf);
        hipHostFree(h_otf);
    }
    if(buf) {
	hipHostUnregister(h_buf);
	hipHostFree(h_buf);
    }
    if(workArea) hipFree(workArea);
    hipProfilerStop();
    hipDeviceReset();
    return retval;
}

int deconv_stream(unsigned int iter, size_t N1, size_t N2, size_t N3, 
                  float *h_image, float *h_psf, float *h_object, float *h_normal) {
    int retval = 0;
    hipfftResult r;
    hipError_t err;
    hipfftHandle planR2C, planC2R;

    hipStream_t fftStream = 0, memStream = 0;

    void *result = 0; // estimated object
    void *buf = 0; // intermediate results
    void *workArea = 0; // cuFFT work area
    hipComplex *h_otf = 0;

    size_t nSpatial = N1*N2*N3; // number of values in spatial domain
    size_t nFreq = N1*N2*(N3/2+1); // number of values in frequency domain
    //size_t nFreq = N1*(N2/2+1); // number of values in frequency domain
    size_t mSpatial, mFreq;
    size_t workSize; // size of cuFFT work area in bytes

    dim3 freqThreadsPerBlock, spatialThreadsPerBlock, freqBlocks, spatialBlocks;

    err = numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock);
    if(err) goto cudaErr;
    err = numBlocksThreads(nFreq, &freqBlocks, &freqThreadsPerBlock);
    if(err) goto cudaErr;

    mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
    mFreq = freqBlocks.x * freqBlocks.y * freqBlocks.z * freqThreadsPerBlock.x * sizeof(hipComplex);

    printf("N: %ld, M: %ld\n", nSpatial, mSpatial);
    printf("Blocks: %d x %d x %d, Threads: %d x %d x %d\n", spatialBlocks.x, spatialBlocks.y, spatialBlocks.z, spatialThreadsPerBlock.x, spatialThreadsPerBlock.y, spatialThreadsPerBlock.z);

    hipDeviceReset();
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    err = hipStreamCreate(&fftStream);
    if(err) goto cudaErr;
    err = hipStreamCreate(&memStream);
    if(err) goto cudaErr;
#if 0
    err = hipEventCreateWithFlags(&memSync, hipEventDisableTiming);
    if(err) goto cudaErr;
#endif

    hipProfilerStart();

    err = hipMalloc(&result, mFreq);
    if(err) goto cudaErr;
    err = hipMalloc(&buf, mFreq); // mFreq > mSpatial
    if(err) goto cudaErr;

    h_otf = (hipComplex*)malloc(nFreq*sizeof(hipComplex));

    printf("Memory allocated.\n");

    err = hipHostRegister(h_image, nSpatial*sizeof(float), 0);
    if(err) goto cudaErr;
    err = hipHostRegister(h_object, nSpatial*sizeof(float), 0);
    if(err) goto cudaErr;
    err = hipHostRegister(h_otf, nFreq*sizeof(hipComplex), 0);
    if(err) goto cudaErr;

    r = createPlans(N1, N2, N3, &planR2C, &planC2R, &workArea, &workSize);
    if(r) goto cufftError;

    r = hipfftSetStream(planR2C, fftStream);
    if(r) goto cufftError;
    r = hipfftSetStream(planC2R, fftStream);
    if(r) goto cufftError;

    printf("Plans created.\n");

    err = hipMemcpyAsync(buf, h_psf, nSpatial*sizeof(float), hipMemcpyHostToDevice, fftStream);
    if(err) goto cudaErr;
    r = hipfftExecR2C(planR2C, (float*)buf, (hipComplex*)buf);
    if(r) goto cufftError;
    err = hipMemcpyAsync(h_otf, buf, nFreq*sizeof(hipComplex), hipMemcpyDeviceToHost, fftStream);

    err = hipStreamSynchronize(fftStream);
    if(err) goto cudaErr;

    printf("OTF generated.\n");

    err = hipMemcpyAsync(result, h_object, nSpatial*sizeof(float), hipMemcpyHostToDevice, fftStream);
    if(err) goto cudaErr;

    for(unsigned int i=0; i < iter; i++) {
        printf("Iteration %d\n", i);
        err = hipMemcpyAsync(buf, h_otf, nFreq*sizeof(hipComplex), hipMemcpyHostToDevice, memStream);
        if(err) goto cudaErr;
        r = hipfftExecR2C(planR2C, (float*)result, (hipComplex*)result);
        if(r) goto cufftError;
	
        hipDeviceSynchronize();

        ComplexMul<<<freqBlocks, freqThreadsPerBlock, 0, fftStream>>>((hipComplex*)result, (hipComplex*)buf, (hipComplex*)result);

        hipDeviceSynchronize();

        err = hipMemcpyAsync(buf, h_image, nSpatial*sizeof(float), hipMemcpyHostToDevice, memStream);
        if(err) goto cudaErr;
        r = hipfftExecC2R(planC2R, (hipComplex*)result, (float*)result);
        if(r) goto cufftError;

        hipDeviceSynchronize();

        FloatDiv<<<spatialBlocks, spatialThreadsPerBlock, 0, fftStream>>>((float*)buf, (float*)result, (float*)result);

        hipDeviceSynchronize();

        err = hipMemcpyAsync(buf, h_otf, nFreq*sizeof(hipComplex), hipMemcpyHostToDevice, memStream);
        if(err) goto cudaErr;
        r = hipfftExecR2C(planR2C, (float*)result, (hipComplex*)result);
        if(r) goto cufftError;

        hipDeviceSynchronize();

        ComplexMul<<<freqBlocks, freqThreadsPerBlock, 0, fftStream>>>((hipComplex*)result, (hipComplex*)buf, (hipComplex*)result);

        hipDeviceSynchronize();

        err = hipMemcpyAsync(buf, h_object, nSpatial*sizeof(float), hipMemcpyHostToDevice, memStream);
        if(err) goto cudaErr;
        r = hipfftExecC2R(planC2R, (hipComplex*)result, (float*)result);
        if(r) goto cufftError;

        hipDeviceSynchronize();

        FloatMul<<<spatialBlocks, spatialThreadsPerBlock, 0, fftStream>>>((float*)buf, (float*)result, (float*)result);

        hipDeviceSynchronize();

        err = hipMemcpyAsync(h_object, result, nSpatial*sizeof(float), hipMemcpyDeviceToHost, fftStream);
        if(err) goto cudaErr;
    }

    hipDeviceSynchronize();

    retval = 0;
    goto cleanup;

cudaErr:
    fprintf(stderr, "CUDA error: %d\n", err);
    retval = err;
    goto cleanup;

cufftError:
    fprintf(stderr, "CuFFT error: %d\n", r);
    retval = r;
    goto cleanup;

cleanup:
    if(fftStream) hipStreamDestroy(fftStream);
    if(memStream) hipStreamDestroy(memStream);
    if(result) hipFree(result);
    if(buf) hipFree(buf);
    if(workArea) hipFree(workArea);
    if(h_otf) {
        hipHostUnregister(h_otf);
        free(h_otf);
    }
    hipHostUnregister(h_image);
    hipHostUnregister(h_object);
    hipProfilerStop();
    hipDeviceReset();
    return retval;
}

hipfftResult createPlans(size_t N1, size_t N2, size_t N3, hipfftHandle *planR2C, hipfftHandle *planC2R, void **workArea, size_t *workSize) {
    hipfftResult r;

    r = hipfftCreate(planR2C);
    if(r) return r;
  	//  r = cufftSetCompatibilityMode(*planR2C, CUFFT_COMPATIBILITY_FFT_PADDING);
  	//  if(r) return r;

    r = hipfftSetAutoAllocation(*planR2C, 0);
    if(r) return r;

    r = hipfftCreate(planC2R);
    if(r) return r;
   	// r = cufftSetCompatibilityMode(*planC2R, CUFFT_COMPATIBILITY_FFT_PADDING);
  	//  if(r) return r;

    r = hipfftSetAutoAllocation(*planC2R, 0);
    if(r) return r;

    size_t tmp;
    r = hipfftGetSize3d(*planR2C, N1, N2, N3, HIPFFT_R2C, workSize);
    //r = hipfftGetSize2d(*planR2C, N1, N2, HIPFFT_R2C, workSize);
    if(r) return r;
    r = hipfftGetSize3d(*planC2R, N1, N2, N3, HIPFFT_R2C, &tmp);
    //r = hipfftGetSize2d(*planC2R, N1, N2, HIPFFT_R2C, &tmp);
    if(r) return r;
    if(tmp > *workSize)
        *workSize = tmp;

    hipError_t err = hipMalloc(workArea, *workSize);
    if(err) return HIPFFT_ALLOC_FAILED;

    r = hipfftSetWorkArea(*planR2C, *workArea);
    if(r) goto error;
    r = hipfftMakePlan3d(*planR2C, N1, N2, N3, HIPFFT_R2C, &tmp);
    //r = hipfftMakePlan2d(*planR2C, N1, N2, HIPFFT_R2C, &tmp);
    if(r) goto error;

    r = hipfftSetWorkArea(*planC2R, *workArea);
    if(r) goto error;
    r = hipfftMakePlan3d(*planC2R, N1, N2, N3, HIPFFT_C2R, &tmp);
    //r = hipfftMakePlan2d(*planC2R, N1, N2, HIPFFT_C2R, &tmp);
    if(r) goto error;

    return HIPFFT_SUCCESS;
error:
    hipFree(*workArea);
    return r;
}

static hipError_t numBlocksThreads(unsigned int N, dim3 *numBlocks, dim3 *threadsPerBlock) {
    unsigned int BLOCKSIZE = 128;
    int Nx, Ny, Nz;
    int device;
    hipError_t err;
    if(N < BLOCKSIZE) {
        numBlocks->x = 1;
        numBlocks->y = 1;
        numBlocks->z = 1;
        threadsPerBlock->x = N;
        threadsPerBlock->y = 1;
        threadsPerBlock->z = 1;
        return hipSuccess;
    }
    threadsPerBlock->x = BLOCKSIZE;
    threadsPerBlock->y = 1;
    threadsPerBlock->z = 1;
    err = hipGetDevice(&device);
    if(err) return err;
    err = hipDeviceGetAttribute(&Nx, hipDeviceAttributeMaxBlockDimX, device);
    if(err) return err;
    err = hipDeviceGetAttribute(&Ny, hipDeviceAttributeMaxBlockDimY, device);
    if(err) return err;
    err = hipDeviceGetAttribute(&Nz, hipDeviceAttributeMaxBlockDimZ, device);
    if(err) return err;
    printf("Nx: %d, Ny: %d, Nz: %d\n", Nx, Ny, Nz);
    unsigned int n = (N-1) / BLOCKSIZE + 1;
    unsigned int x = (n-1) / (Ny*Nz) + 1;
    unsigned int y = (n-1) / (x*Nz) + 1;
    unsigned int z = (n-1) / (x*y) + 1;
    if(x > Nx || y > Ny || z > Nz) {
        return hipErrorInvalidConfiguration;
    }
    numBlocks->x = x;
    numBlocks->y = y;
    numBlocks->z = z;

    return hipSuccess;
}


int conv_device(size_t N1, size_t N2, size_t N3, 
                  float *h_image, float *h_psf, float *h_out, unsigned int correlate) {

    int retval = 0;
    hipfftResult r;
    hipError_t err;
    hipfftHandle planR2C, planC2R;

	std::cout<<"Starting Cuda convolution\n";
	printf("input size: %d %d %d", N1, N2, N3);

    float *image = 0; // convolved image (constant)
    float *psf=0;
	float *out = 0; // estimated object
	
    hipComplex *otf = 0; // Fourier transform of PSF (constant)
    void *buf = 0; // intermediate results
    void *workArea = 0; // cuFFT work area

    size_t nSpatial = N1*N2*N3; // number of values in spatial domain
    size_t nFreq = N1*N2*(N3/2+1); // number of values in frequency domain
    //size_t nFreq = N1*(N2/2+1); // number of values in frequency domain
    size_t mSpatial, mFreq;

    dim3 freqThreadsPerBlock, spatialThreadsPerBlock, freqBlocks, spatialBlocks;
    size_t workSize; // size of cuFFT work area in bytes

    err = numBlocksThreads(nSpatial, &spatialBlocks, &spatialThreadsPerBlock);
    if(err) goto cudaErr;
    err = numBlocksThreads(nFreq, &freqBlocks, &freqThreadsPerBlock);
    if(err) goto cudaErr;

    mSpatial = spatialBlocks.x * spatialBlocks.y * spatialBlocks.z * spatialThreadsPerBlock.x * sizeof(float);
    mFreq = freqBlocks.x * freqBlocks.y * freqBlocks.z * freqThreadsPerBlock.x * sizeof(hipComplex);

    printf("N: %ld, M: %ld\n", nSpatial, mSpatial);
    printf("Blocks: %d x %d x %d, Threads: %d x %d x %d\n", spatialBlocks.x, spatialBlocks.y, spatialBlocks.z, spatialThreadsPerBlock.x, spatialThreadsPerBlock.y, spatialThreadsPerBlock.z);
	fflush(stdin);

	std::cout<<"N: "<<nSpatial<<" M: "<<mSpatial<<"\n"<<std::flush;
	std::cout<<"Blocks: "<<spatialBlocks.x<<" x "<<spatialBlocks.y<<" x "<<spatialBlocks.z<<", Threads: "<<spatialThreadsPerBlock.x<<" x "<<spatialThreadsPerBlock.y<<" x "<<spatialThreadsPerBlock.z<<"\n";
    
	hipDeviceReset();

    hipProfilerStart();

    err = hipMalloc(&image, mSpatial);
    if(err) goto cudaErr;
    err = hipMalloc(&out, mSpatial);
    if(err) goto cudaErr;
	err = hipMalloc(&psf, mSpatial);
    if(err) goto cudaErr;
	
    err = hipMalloc(&buf, mFreq); // mFreq > mSpatial
    if(err) goto cudaErr;

	err = hipMalloc(&otf, mFreq); // mFreq > mSpatial
    if(err) goto cudaErr;

    err = hipMemset(image, 0, mSpatial);
    if(err) goto cudaErr;
    err = hipMemset(out, 0, mSpatial);
    if(err) goto cudaErr;

    err = hipMemcpy(image, h_image, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;
    err = hipMemcpy(out, h_out, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;

    err = hipMemcpy(psf, h_psf, nSpatial*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;

    // BN it looks like this function was originall written for the array organization used in matlab.  I Changed the order of the dimensions
    // to be compatible with imglib2 (java). TODO - add param for array organization 
    r = createPlans(N1, N2, N3, &planR2C, &planC2R, &workArea, &workSize);
    if(r) goto cufftError;

    r = hipfftExecR2C(planR2C, psf, otf);
    if(r) goto cufftError;

    // BN flush the buffer for debugging in Java.
    fflush(stdout);
    
	r = hipfftExecR2C(planR2C, image, (hipfftComplex*)buf);
    if(r) goto cufftError;
    
	if (correlate==1) {
		ComplexConjugateMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
	}
	else {
		ComplexMul<<<freqBlocks, freqThreadsPerBlock>>>((hipComplex*)buf, otf, (hipComplex*)buf);
	}        

	r = hipfftExecC2R(planC2R, (hipfftComplex*)buf, (float*)out);
    if(r) goto cufftError;
	

		FloatDivByConstant<<<spatialBlocks, spatialThreadsPerBlock>>>((float*)out,(float)nSpatial);
    
		err = hipMemcpy(h_out, out, nSpatial*sizeof(float), hipMemcpyDeviceToHost);
    
		retval = 0;
    goto cleanup;

cudaErr:
    fprintf(stderr, "CUDA error: %d\n", err);
    retval = err;
    goto cleanup;

cufftError:
    fprintf(stderr, "CuFFT error: %d\n", r);
    retval = r;
    goto cleanup;

cleanup:
    if(image) hipFree(image);
    if(out) hipFree(out);
    if(otf) hipFree(otf);
    if(buf) hipFree(buf);
    if(workArea) hipFree(workArea);
    hipProfilerStop();
    hipDeviceReset();
    return retval;
}

